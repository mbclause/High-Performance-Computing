// Project 2 [Parallelized Shear Sort]
// Michael Clausen, Jayson Doty, Youssef Hegazy
// Professor Doina Bein
// CPSC 479
// File: Project2.cu
// 
// Description: Program to sort an n x n matrix using Parallel Shear Sort. It is
// implimented in CUDA. The user inputs the value of n and the n x n matrix is 
// filled with random values ranging from -100 to 900. This
// matrix is then displayed. The sorting is then
// accomplished in two stages. In the Odd Stage, each odd row is sorted
// from left to right and each even row is sorted from right to left.
// In the Even Stage, each column is sorted from top to bottom.
// The sorting of the rows and columns is done using Bubble Sort.
// The resulting matrix is sorted in a "snake like" pattern and 
// then displayed.

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>

using namespace std;



/*
Function: ShearSortRow
Description: Kernel function to sort each row of the matrix using Bubble Sort.
Odd rows are sorted from left to right and odd rows are sorted from right to left.
*/
__global__ void ShearSortRow(int* matrix, int n) 
{
    // calculate thread ID
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    // check that the thread ID is in bounds
    if (threadID < n) 
    {
        // loop over the matrix rows
        for (int i = 0; i < n; i++) 
        {
            // loop over the matrix columns
            for (int j = 0; j < n - 1; j++) 
            {
                // Calculate the index in the matrix based on the threadID and column index
                int index = threadID * n + j;

                // odd rows swap so that they are sorted left to right
                // even rows swap so that they are sorted right to left
                if ((threadID % 2 == 0 && matrix[index] > matrix[index + 1]) ||
                    (threadID % 2 == 1 && matrix[index] < matrix[index + 1])) 
                {
                    // Swap
                    int temp = matrix[index];
                    matrix[index] = matrix[index + 1];
                    matrix[index + 1] = temp;
                }
            }
        }
    }
} // end "ShearSortRow"



/*
Function: ShearSortColumn
Description: Kernel function to sort each column of the matrix using Bubble Sort.
Columns are sorted normally, from top to bottom.
*/
__global__ void ShearSortColumn(int* matrix, int n) 
{
    // calculate thread ID
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    // check that the thread ID is in bounds
    if (threadID < n) 
    {
        // loop through matrix rows
        for (int i = 0; i < n; i++) 
        {
            // loop through matrix columns
            for (int j = 0; j < n - 1; j++) 
            {
                // Calculate the index in the matrix based on the column index and threadID
                int index = j * n + threadID;

                // compare and swap
                if (matrix[index] > matrix[index + n]) 
                {
                    // Swap
                    int temp = matrix[index];
                    matrix[index] = matrix[index + n];
                    matrix[index + n] = temp;
                }
            }
        }
    }
} // end "ShearSortColumn"



/*
Function: main
*/
int main() 
{
    int N = 0;

    // get value of n from the user
    cout << "Enter an integer value for n greater than 0: ";

    cin >> N;

    // check that n is positive
    while(N < 1)
    {
        cout << "n must be greater than 0. Re-enter value: ";

        cin >> N;
    }

    int ha[N][N], *dm;

    // Initialize the matrix with random values, and print.
    std::cout<<"Matrix before sorting: "<<std::endl;
    std::cout<<"------------------------------------------------"<<std::endl;

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++) 
        {
            ha[i][j] = (rand() % 1000) - 100;
            std::cout<<setw(5)<<ha[i][j]<<" ";
        }
        std::cout<<std::endl;
    }
    std::cout<<"------------------------------------------------"<<std::endl;

    // allocate matrix on the GPU and copy over the matrix initialized on the CPU
    hipMalloc(&dm, N * N * sizeof(int));
    hipMemcpy(dm, ha, N * N * sizeof(int), hipMemcpyHostToDevice);
    
    // kernel dimensions: n threads per block, and 1 block per grid
    dim3 threadsPerBlock(N);
    dim3 numBlocks(1);

    // sort rows in odd stage and columns in even stage log(n) times
    // synchronizing the device after each stage
    for (int i = 0; i < log2(N) + 1; i++) 
    {
        ShearSortRow<<<numBlocks, threadsPerBlock>>>(dm, N);
        hipDeviceSynchronize();
        ShearSortColumn<<<numBlocks, threadsPerBlock>>>(dm, N);
        hipDeviceSynchronize();
    }
    
    // Copy the matrix back to the host
    hipMemcpy(ha, dm, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(dm);

    // Print the sorted matrix
    std::cout<<"Matrix After sorting: "<<std::endl;
    std::cout<<"------------------------------------------------"<<std::endl;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++) 
        {
            std::cout<<setw(5)<<ha[i][j]<<" ";
        }
        std::cout<<std::endl;
    }
    std::cout<<"------------------------------------------------"<<std::endl;
    return 0;
} // end "main"
