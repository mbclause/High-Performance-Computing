/*
 Change the size of array[] to 8000 and do what Problem 1 and
Problem 2 ask. Check if you are still able to do what Problem 1 and Problem 2 ask
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;








__global__ void Init_Array()
{

        const int size = 8000;

        int  array[size];

        for(int i = 0; i < size; i++)
        {
                array[i] = 2;
        }

        for(int i = 0; i < size; i++)
        {
                printf("%d ", array[i]);

                if(i % 50 == 0)
                        printf("\n");
        }

        printf("\n");
}



int main()
{

        Init_Array<<<1, 1>>>();

        hipDeviceSynchronize();

        return 0;

}