/*
Change the size of array[] to 8000 and do what Problem 1 and Problem 2 ask. Check if you are still able to do what Problem 1 and Problem 2 asks.
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;



// initialze the array using the gpu
__global__ void Init_Array(int *array, const int size)
{
        int i = threadIdx.x;

        if(i < size)
                array[i] = 2;
}



int main()
{
	// size of array and number of threads is 8000
        const int size = 8000;

        int  *deviceArray;

        int  cpuArray[size];

	// allocate array memory on GPU
        hipMalloc((void **)&deviceArray, size * sizeof(int));

        dim3   blockSize(size);

        Init_Array<<<1, blockSize>>>(deviceArray, size);

	// copy gpu array to cpu array
        hipMemcpy(cpuArray, deviceArray, size * sizeof(int), hipMemcpyDeviceToHost);

	// display array, because 8000 is too many threads, array values will not be initialize to 2
        for(int i = 0; i < size; i++)
                cout << cpuArray[i] << " ";

        cout << endl;

        hipFree(deviceArray);

        return 0;

}