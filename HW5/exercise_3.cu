/*
 Create another kernel that adds i to array[i] and do what Problem 1
and Problem 2 ask.
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;




__device__  void  Add_Array(int array[], int size)
{
        for(int i = 0; i < size; i++)
        {
                array[i] += i;
        }

        for(int i = 0; i < size; i++)
        {
                printf("%d ", array[i]);

                if(i % 50 == 0)
                        printf("\n");
        }

        printf("\n");
}





__global__ void Init_Array()
{

        const int size = 1024;

        int  array[size];

        for(int i = 0; i < size; i++)
        {
                array[i] = 2;
        }

        Add_Array(array, size);
}



int main()
{

        Init_Array<<<1, 1>>>();

        hipDeviceSynchronize();

        return 0;

}