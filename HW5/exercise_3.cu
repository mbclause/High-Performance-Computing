/*
Create another kernel that adds i to array[i] and do what Problem 1 and Problem 2 ask.
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;



// initialize the array using the GPU
__global__ void Init_Array(int *array, const int size)
{
        int i = threadIdx.x;

        if(i < size)
                array[i] = 2;
}




// add index to all array values using the GPU
__global__  void  Add_Array(int  *array, const int size)
{

        int i = threadIdx.x;

        if(i < size)
                array[i] += i;

}




int main()
{
	// size of the array and the number of threads
        const int size = 1024;

        int  *deviceArray;

        int  cpuArray[size];

	// allocate memory on the GPU for the array
        hipMalloc((void **)&deviceArray, size * sizeof(int));

        dim3   blockSize(size);

	// initialize array on GPU
        Init_Array<<<1, blockSize>>>(deviceArray, size);

	// add index to array on GPU
        Add_Array<<<1, blockSize>>>(deviceArray, size);

	// copy array from the GPU to CPU
        hipMemcpy(cpuArray, deviceArray, size * sizeof(int), hipMemcpyDeviceToHost);

	// display array
        for(int i = 0; i < size; i++)
                cout << cpuArray[i] << " ";

        cout << endl;

        hipFree(deviceArray);

        return 0;

}