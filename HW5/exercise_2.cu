/*
Change the size of array[] to 1024 and do what Problem 1 asks.
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;



// function to initialize the array using the GPU
__global__ void Init_Array(int *array, const int size)
{
        int i = threadIdx.x;

        if(i < size)
                array[i] = 2;
}





int main()
{
	// the size of the array and thus the number of GPU threads
        const int size = 1024;

        int  *deviceArray;

        int  cpuArray[size];

	// allocate memory for the GPU array
        hipMalloc((void **)&deviceArray, size * sizeof(int));

        dim3   blockSize(size);

        Init_Array<<<1, blockSize>>>(deviceArray, size);

	// copy the array from the GPU to the CPU
        hipMemcpy(cpuArray, deviceArray, size * sizeof(int), hipMemcpyDeviceToHost);

	// display the array
        for(int i = 0; i < size; i++)
                cout << cpuArray[i] << " ";

        cout << endl;

        hipFree(deviceArray);

        return 0;

}