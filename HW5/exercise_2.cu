/*
 Change the size of array[] to 1024 and do what Problem 1 asks
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;



__global__ void Init_Array()
{

        const int size = 1024;

        int  array[size];

        for(int i = 0; i < size; i++)
        {
                array[i] = 2;
        }

        for(int i = 0; i < size; i++)
        {
                printf("%d ", array[i]);

                if(i % 50 == 0)
                        printf("\n");
        }

        printf("\n");
}




int main()
{

        Init_Array<<<1, 1>>>();

        hipDeviceSynchronize();

        return 0;

}