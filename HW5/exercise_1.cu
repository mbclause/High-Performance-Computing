/*
Write a CUDA program to initialize an array array[] of size 64 to all 2s
in parallel.
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;



__global__ void Init_Array()
{

        const int size = 64;

        int  array[size];

        for(int i = 0; i < size; i++)
        {
                array[i] = 2;
        }

        for(int i = 0; i < size; i++)
        {
                printf("%d ", array[i]);
        }

        printf("\n");
}




int main()
{

        Init_Array<<<1, 5>>>();

        hipDeviceSynchronize();

        return 0;

}