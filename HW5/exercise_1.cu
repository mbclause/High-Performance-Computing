/*
Write a CUDA program to initialize an array array[] of size 64 to all 2s
in parallel.
*/



#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;



// Function to initialize the array using the GPU
__global__ void Init_Array(int *array, const int size)
{
        int i = threadIdx.x;

        if(i < size)
                array[i] = 2;
}







int main()
{
	// the size of the array, and thus the number of GPU threads
        const int size = 64;

        int  *deviceArray;

        int  cpuArray[size];

	// allocate memory for the device array
        hipMalloc((void **)&deviceArray, size * sizeof(int));

	// set the number of threads
        dim3   blockSize(size);

        Init_Array<<<1, blockSize>>>(deviceArray, size);
	
	// copy the array initialized on the GPU to the cpu array
        hipMemcpy(cpuArray, deviceArray, size * sizeof(int), hipMemcpyDeviceToHost);

	// display the array contents
        for(int i = 0; i < size; i++)
                cout << cpuArray[i] << " ";

        cout << endl;

        hipFree(deviceArray);

        return 0;

}
