/*
File: exercise_2.cu
Description: This CUDA program uses a global function to compute the square of a given NxN matrix, N being 16.
For example, if the matrix given is A with 16x16 elements, the host function will compute the matrix AxA.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;


#define N 16



/*
Function: InitMatrix
Description: A global function which initializes an unsigned matrix.
*/
__global__ void InitMatrix(unsigned *matrix)
{

        unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

        matrix[id] = id;

} // end "InitMatrix"



/*
Function: square
Description: Takes an NxN matrix A and computes its square AxA.
*/
__global__ void square ( unsigned *matrix, unsigned *result, unsigned matrixsize)
{

        unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

        unsigned ii = id / matrixsize;

        unsigned jj = id % matrixsize;

        for (unsigned kk = 0; kk < matrixsize; ++kk)
        {

                result[ii * matrixsize + jj] += matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
        }

} // end "square"



/*
main
*/
int main()
{

        unsigned *matrix, *hmatrix, *resultMatrix;

        int matrixSize = N;

        hipMalloc(&matrix, N * N * sizeof(unsigned));

        hipMalloc(&resultMatrix, N * N * sizeof(unsigned));

        hmatrix = (unsigned *)malloc(N * N * sizeof(unsigned));

        InitMatrix<<<N, N>>>(matrix);

        hipMemcpy(hmatrix, matrix, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);

        printf("Original Matrix\n");

        // display the original matrix
        for (unsigned ii = 0; ii < N; ++ii)
        {

                for (unsigned jj = 0; jj < N; ++jj)
                {

                        printf("%2d ", hmatrix[ii * N + jj]);

                }

                printf("\n");

        }

        square<<<N, N>>>(matrix, resultMatrix, matrixSize);

        hipMemcpy(hmatrix, resultMatrix, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);

        printf("\nSquared Matrix\n");

        // display the squared matrix
        for (unsigned ii = 0; ii < N; ++ii)
        {

                for (unsigned jj = 0; jj < N; ++jj)
                {

                        printf("%2d ", hmatrix[ii * N + jj]);

                }

                printf("\n");

        }

        return 0;

} // end "main"