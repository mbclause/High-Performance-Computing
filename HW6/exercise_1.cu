/*
File:
Description: This CUDA program uses a global function to compute the number of even values in a given NxN matrix, N being 16.
*/



#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

#define SIZE N * N

#define BLOCKSIZE 4



/*
Function: isEven
Description: A device function that checks if a value is even or odd. Returns 1 or 0 respectively.
*/
__device__ int IsEven(int val)
{

        if (val % 2 == 0) return 1;

        return 0;

} // end "isEven"



/*
Function: compute
Description: A global function that keeps track of the number of even values in its matrix/array block by storing
1 or zero in d_out depending on if the respective element is even or not.
*/
__global__ void compute(int *d_in,int *d_out) {

        d_out[threadIdx.x] = 0;

        for (int i=0; i<SIZE/BLOCKSIZE;i++)
        {

                int val = d_in[i*BLOCKSIZE + threadIdx.x];

                d_out[threadIdx.x] += IsEven(val);

        }

} // end "compute"



/*
Function: outer_compute
Description: Host function which launches size/blocksize kernels and launches them.
It copies the resulting array from the gpu to the cpu, stored in h_out_array.
*/
__host__ void outer_compute (int *h_in_array, int *h_out_array)
{

        int *d_in_array, *d_out_array;

        hipMalloc((void **) &d_in_array,SIZE*sizeof(int));

        hipMalloc((void **) &d_out_array,SIZE*sizeof(int));

        hipMemcpy(d_in_array, h_in_array,SIZE*sizeof(int),hipMemcpyHostToDevice);

        compute<<<1, BLOCKSIZE, 0>>>(d_in_array, d_out_array);

        hipMemcpy(h_out_array, d_out_array,SIZE*sizeof(int),hipMemcpyDeviceToHost);

} // end "outer_compute"



/*
Function: main
*/
int main()
{

        int *in_array, *out_array;

        int sum = 0;

        in_array = (int *)malloc(SIZE * sizeof(int));

        out_array = (int *)malloc(SIZE * sizeof(int));

        // initialize matrix
        for(int i = 0; i < SIZE; i++)
                in_array[i] = i;

        printf("Matrix\n");

        // display the matrix
        for(int i = 0; i < N; i++)
        {
                for(int j = 0; j < N; j++)
                {
                        printf("%2d ", in_array[i * N + j]);
                }

                printf("\n");
        }

        outer_compute(in_array, out_array);

        // add up the total number of even values from the output array
        for (int i=0; i<SIZE; i++)
        {

                sum+=out_array[i];

        }

        printf ("\nNumber of even values = %d\n",sum);

} // end "main"